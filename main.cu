#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#include "vec3.h"
#include "colour.h"
#include "ray.h"
#include "sphere.h"
#include "hittable_list.h"
#include "camera.h"
#include "material.h"


/** 
 * return error if a CUDA error if encountered during runtime
 * 
 * @param[in] val the error value of a CUDA function call
 * 
 * @note checkCudaErrors() used as simplifcation of check_cuda()
 * @note see hipError_t enum section of https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
 *      for error code documentation
 */
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__)

void check_cuda(cudaError_T result, char const *const func, const char *const file, int const line) {
    
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ";" << line << " '" << func << "' \n";

        // make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}


/**
 * determine colour of objects hit by a ray to a max 50 depth
 * 
 * the steps are: (1) calculate ray from eye to pixel, (2) determine objects ray intersects, &
 * (3) compute a colour for that intersection point
 * 
 * @param[in] r the ray being shot out from the eye
 * @param[in] world the series of objects in the scene
 * @param[in] local_rand_state the CUDA random state
 * 
 * @returns colour where ray intersects with an object
 * 
 * @relatesalso ray
 */
__device__ colour ray_colour(const ray& r, hittable **world, hiprandState *local_rand_state) {
    
    ray cur_ray = r;
    float cur_attenuation = 1.0f;
    for (int i = 0; i < 50 ; i++) {
        hit_record rec;
        if ((*world)->hit(r, 0.001f, FLT_MAX, rec)) {
            vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
            cur_attenuation *= 0.5f
            cur_ray = ray(rec.p, target-rec.p)
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            colour c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }

    return colour(0.0, 0.0, 0.0)    // exceeded recursion depth of 50 
}


/**
 * computes random initialization for the renderer
 * 
 * @param[in] max_x the image width in pixels
 * @param[in] max_y the image height in pixels
 * @param[out] rand_state the CUDA random state
 * 
 * @note can also be initialized at the top of render() based on preference
*/
__global__ render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j*max_x + i;

    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);    
}


/**
 * identify coordinates of each thread in the image (i, j) and writes it to fb[]
 * 
 * @param[out] fb the frame buffer used to store image colour data
 * @param[in] max_x the image width in pixels
 * @param[in] max_y the image height in pixels
 * @param[in] samples the number of samples per pixel
 * @param[in] cam the camera where rays are shot from
 * @param[in] world the series of objects in the scene
 * @param[in] rand_state the CUDA random state
 * 
 * @warning fb should be hipMallocManaged()
 */
__global__ void render(colour *fb, int max_x, int max_y, 
                      int samples, camera **cam, hittable **world, hiprandState *rand_state) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;

    int pixel_index = y*max_x + x;

    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);

    for(int s=0; s < samples; s++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        col += ray_colour(r, world);
    }
    
    fb[pixel_index] = col/float(samples);
}


/// initializes the scene and a list of objecs in the scene
__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    // ensures function only runs once in kernal
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0,0,-1), 0.5,
                               new lambertian(vec3(0.8, 0.3, 0.3)));
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100,
                               new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               new metal(vec3(0.8, 0.6, 0.2), 1.0));
        d_list[3] = new sphere(vec3(-1,0,-1), 0.5,
                                 new metal(vec3(0.8, 0.8, 0.8), 0.3));
        *d_world  = new hitable_list(d_list,4);
        *d_camera = new camera();
    }
}


/// deletes the scene and objects inside
__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    delete *d_world;
    delete *d_list;
    delete *d_camera;
}


int main() {

    // image
    const int image_x = 1440;             // image width
    const int image_y = 720;              // image height
    const int image_s = 100;              // image samples
    const int thread_x = 8;               // thread block x dimension
    const int thread_y = 8;               // thread block y dimension

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << thread_x << "x" << thread_y << " blocks.\n";

    int image_pixels = image_x * image_y;
    
    // allocate frame buffer (FB) on host to hold RGB values for GPU-CPU communication
    colour *fb;
    size_t fb_size = 3 * image_pixels * sizeof(colour);             // each pixel contains 3 float values (RGB)
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));       // typecast &fb as void** due to CUDA documentation

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMallocManaged((void **)&d_rand_state, image_pixels*sizeof(hiprandState)));

    // initializes environment objects
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 2*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));

    create_world<<<1,1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());                       // tells CPU to wait until kernal is done before beginning render

    // render FB
    clock_t start, stop;
    start = clock();

    dim3 blocks(image_x / thread_x + 1, image_y / thread_y + 1);    // blocks needed is total image pixels / threads per block
    dim3 threads(thread_x, thread_y);                               // thread_x * thread_y threads in a single block

    render_init<<<blocks, threads>>>(fb, image_x, image_y, image_s, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize()); 

    render<<<blocks, threads>>>(fb, image_x, image_y, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " s.\n";

    // output FB as image
    std::cout << "P3\n" << image_x << ' ' << image_y << "\n255\n";

    // iterate through FB elements in intervals of 3
    for (int j = image_y-1; j >= 0; j--) {
        for (int i = 0; i < image_x; i++) {
            size_t pixel_index = j*image_x + i;
            write_color(std::cout, fb[pixel_index], image_s);
        }
    }

    // clean up
    free_world<<<1,1>>>(d_list, d_world, d_camera );
    checkCudaErrors(hipDeviceSynchronize());                       // ensure all kernal processes are done before cleaning up
    checkCudaErrors(hipFree(fb));                                  // free FB memory

    hipDeviceReset();                                              // useful for cuda-memcheck --leak-check full
}
